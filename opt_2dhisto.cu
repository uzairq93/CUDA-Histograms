#include "hip/hip_runtime.h"
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <stdio.h>

#include <cutil.h>
#include "util.h"
#include "ref_2dhisto.h"

#define THREAD_DIM_X 32
#define BLOCK_DIM_X 32
#define BLOCK_DIM_Y 32
#define PAD_WIDTH ((INPUT_WIDTH + 128) & 0xFFFFFF80) // See alloc_2d in util.cpp
#define BIN_COUNT HISTO_HEIGHT*HISTO_WIDTH
#define INPUT_COUNT INPUT_HEIGHT*INPUT_WIDTH
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

/* Start of my device functions. All functions to be executed on the device must go here, above opt_2dhisto(), because they won't 
   be included in the corresponding header file. */
__global__ void Baseline_Kernel(uint32_t *input, uint32_t *kernel_bins){
    /* Blocks were 32x32 when evaluating this kernel, so memory accesses were coalesced */
    __shared__ uint32_t sub_hist[BIN_COUNT];
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    /* STEP 1: Initialize your assigned bins inside of shared memory
                There are no bank conflicts here because each warp edits a unique offset
    */
    for (int pos = threadIdx.x; pos < BIN_COUNT; pos += blockDim.x)
        sub_hist[pos] = 0;
    __syncthreads();

    /* STEP 2: Add your element to the block's sub histogram
                There may be bank conflicts when writing to shared memory
                This must be written in a for loop if thread coarsening is desired */
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH)
    {
        uint32_t index = input[col + row * PAD_WIDTH];

        // The Always_AtomicAdd kernel just removes this if statement
        if (kernel_bins[index] < 255)
            atomicAdd(kernel_bins + index, 1);
    }
    __syncthreads();

    /* STEP 3: Merge your results with global memory
                There are no bank conflicts here because each warp reads a continuous 32-integer slot
    */
    for (int pos = threadIdx.x; pos < BIN_COUNT; pos += blockDim.x)
    {
        atomicAdd(kernel_bins + pos, sub_hist[pos]);
        if (kernel_bins[pos] > 255)
            atomicExch(kernel_bins + pos, 255);
    }
}


__global__ void NoSharedMem_Kernel(uint32_t *input, uint32_t *kernel_bins) {
    /* Blocks were 32x32 when evaluating this kernel for runtime, so memory accesses were coalesced */
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;

    if (row < INPUT_HEIGHT && col < INPUT_WIDTH)
    {
        uint32_t index = input[col + row * PAD_WIDTH];

        // The Always_AtomicAdd kernel just removes this if statement
        if (kernel_bins[index] < 255) 
            atomicAdd(kernel_bins + index, 1);
    }
}
__global__ void Shuffle(uint32_t *input, uint32_t *shuffle_input) {
    /* Blocks were 32x32 when evaluating this kernel for runtime, so memory accesses were coalesced */
    int col = blockDim.x * blockIdx.x + threadIdx.x;
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    __shared__ uint32_t temp[32][32];
    //first, shuffle the input
    if (row < INPUT_HEIGHT && col < PAD_WIDTH)
    {
        
        temp[threadIdx.x][threadIdx.y] = input[col + row * PAD_WIDTH];
    }
    __syncthreads();
    shuffle_input[col+row*PAD_WIDTH] = temp[threadIdx.y][threadIdx.x];
    __syncthreads();
    /*
    //then increment bins
    if (row < INPUT_HEIGHT && col < INPUT_WIDTH)
    {
        uint32_t index = input[col + row * PAD_WIDTH];

        // The Always_AtomicAdd kernel just removes this if statement
        if (kernel_bins[index] < 255) 
            atomicAdd(kernel_bins + index, 1);
    }*/
}
__global__ void TLPNoSharedMem_Kernel(uint32_t *input, uint32_t *kernel_bins)
{
    // NOTE: This is failing because I'm including padded spaces as bins, but it still can
    //       show off the time to say we tried in in the report. 

    /* Threads are covering 32 elements each, so memory accesses are coalesced */
    int iblock = blockIdx.x + blockIdx.y * gridDim.x;
    int index = threadIdx.x + 2 * iblock * gridDim.x;

    uint32_t element1 = input[index];
    uint32_t element2 = input[index + blockDim.x];
    uint32_t element3 = input[index + 2 * blockDim.x];

    // The Always_AtomicAdd kernel just removes this if statement
    if (kernel_bins[element1] < 255)
        atomicAdd(kernel_bins + element1, 1);
    if (kernel_bins[element2] < 255)
        atomicAdd(kernel_bins + element2, 1);
    if (kernel_bins[element2] < 255)
        atomicAdd(kernel_bins + element3, 1);
}

/* End of my device functions */

/* The following function was included for us, and must keep this name. We can call it with any set of parameters our kernels and 
   optimizations may need. However, all memory allocations and transfers must be done outside of this function. This function 
   should only contain a call to the GPU histogramming kernel. */
//needed to remove shuffling from opt2d
void opt_2dhisto(uint32_t *dev_input, uint32_t *dev_kernel_bins/*, uint32_t *shuffle_input*/)
{
    hipMemset(dev_kernel_bins, 0, BIN_COUNT * sizeof(uint32_t)); // Reset bins every time so the timing works properly
    
    dim3 gridDims(MAX(PAD_WIDTH / BLOCK_DIM_X, 1), MAX(INPUT_HEIGHT / BLOCK_DIM_Y, 1), 1);
    dim3 blockDims(BLOCK_DIM_X, BLOCK_DIM_Y, 1);
    //Shuffle<<<gridDims, blockDims>>>(dev_input, shuffle_input);

    //hipDeviceSynchronize();
    Baseline_Kernel<<<gridDims, blockDims>>>(dev_input, dev_kernel_bins);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) printf("Error: %s\n", hipGetErrorString(err));
}

/* The functions below were written explicitly by Kyle Williams, and are called in test_harness.cpp for setup/teardown of the 
   GPU histogramming kernel. That file is to be compiled with gcc, so we need all calls to CUDA functions to be contained here 
   instead. */

uint32_t* allocate_input_on_device() {
    uint32_t* device_input;
    hipMalloc((void **)&device_input, INPUT_HEIGHT * PAD_WIDTH * sizeof(uint32_t));
    return device_input;
  
}

//Need to allocate space for the shuffled input, REMOVE IF NOT SHUFFLING
uint32_t* allocate_bins_on_device() {
    uint32_t* device_bins;
    hipMalloc((void **)&device_bins, BIN_COUNT * sizeof(uint32_t));
    return device_bins;
}

void copy_input_and_initialize_bins(uint32_t **input, uint32_t *device_input, uint32_t *device_bins) {
    /* STEP 1: Copy the elements from the input into the device input */
    for (int i = 0; i < INPUT_HEIGHT; i++) {
        int offset = i * PAD_WIDTH;
        hipMemcpy(&device_input[offset], input[i], PAD_WIDTH * sizeof(uint32_t), hipMemcpyHostToDevice);
    }
    /* STEP 3: Initialize the bins to 0 */
    hipMemset(device_bins, 0, BIN_COUNT * sizeof(uint32_t));
}

void transfer_bins(uint8_t* output_bins, uint32_t* device_bins) {
    /* STEP 1: Transfer the computed frequencies in device_bins to a copy array in local memory */
    uint32_t kernel_bins_copy[BIN_COUNT];
    hipMemcpy(&kernel_bins_copy, device_bins, BIN_COUNT * sizeof(uint32_t), hipMemcpyDeviceToHost);

    /* STEP 2: Transfer those computed frequencies back to the expected uint8_t object. Account for 
               overflow and cap frequencies at 255 */
    for (int i = 0; i < BIN_COUNT; i++) {
        if (kernel_bins_copy[i] > 255) output_bins[i] = 255;
        else output_bins[i] = kernel_bins_copy[i];
    }
}
void free_device_memory(uint32_t *device_input, uint32_t *device_bins) {
    hipFree(device_input);
    hipFree(device_bins);
}
void free_shuffle_memory(uint32_t *shuffle_input) {
    hipFree(shuffle_input);   
}
